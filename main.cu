#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include "posterize.h"
#include "serialposterize.h"

int main(int argc, char **argv)
{
  //uchar4 *h_image, *d_image;
  char* input_file;
  char* output_file;
  char* command;
  int colors;
  if (argc < 4) {
    printf("Provide command to run, input, and output files.\n");
    exit(1);
  }
  if (argc == 5) {
    colors = atoi(argv[3]);
  } else {
    colors = 6;
  }
  input_file = argv[1];
  output_file = argv[2];
  command = argv[3];
  IplImage* img = cvLoadImage(input_file, CV_LOAD_IMAGE_COLOR);
  IplImage* out_img = cvCreateImage(cvGetSize(img), img->depth, img->nChannels);
  cvCopy(img, out_img, NULL);
  size_t cols = img->width;
  size_t rows = img->height;
  int channels = img->nChannels;
  char* image_rgb;
  image_rgb = img->imageData;
  char* out_image_rgb;
  if (strcmp(command,"serial-posterize")) {
    out_image_rgb = processSerialPosterize(image_rgb, cols, rows, channels, colors);
  } else if (strcmp(command, "posterize")) {
    out_image_rgb = processPosterize(image_rgb, cols, rows, channels, colors);
  } else {
    printf("Command '%s' is not valid", command);
    exit(1);
  }
  out_img->imageData = out_image_rgb;
  int p[3];
  p[0] = CV_IMWRITE_JPEG_QUALITY;
  p[1] = 95;
  p[2] = 0;
  cvSaveImage(output_file, out_img, p);
  cvReleaseImage(&img);
  return 0;
}