#include "hip/hip_runtime.h"
#include "utils.h"
#include "mode.h"

__global__
void mode(const unsigned char* input,
          unsigned char* output,
          size_t cols, size_t rows, int channels)
{
  int x = blockDim.x*blockIdx.x+threadIdx.x;
  int y = blockDim.y*blockIdx.y+threadIdx.y;
  if (x >= cols || y >= rows) {
      return;
  }

  int idx = y*cols+x;
  int dim = 9;
  int offset, Offset = 0;
  int count = 0, maxCount = 0;
  unsigned char mode = NULL;
  int i, j, k, J, K = 0;

  // for each channel...
  for (i = 0; i < channels; i++) {
    // for every pixel per channel...
    for (j = -dim/2; j <= dim/2; j++){
      count = 0;
      for (k = -dim/2; k <= dim/2; k++) {
        offset = idx+cols*j+k;
        if ((x+k >= cols || y+j >= rows) || (x+k < 0 || y+j < 0)) {
          continue;
        }

        // compare it to every other pixel
        for (J = -dim/2; J <= dim/2; J++){
          for (K = -dim/2; K <= dim/2; K++) {
            Offset = idx+cols*J+K;
            if ((x+K >= cols || y+J >= rows) || (x+K < 0 || y+J < 0)) {
              continue;
            }

            if (input[offset*channels + i] == input[Offset*channels + i]) {
              count++;
            }
          }
        }
        if (count > maxCount) {
          maxCount = count;
          mode = input[offset*channels + i];
        }
      }
    }

    if (maxCount > 1) {
      output[idx*channels + i] = mode;
    }
    else {
      output[idx*channels + i] = input[idx*channels + i];
    }
    maxCount = 0;
  }
}

char* processMode(char* image_rgb, size_t cols, size_t rows, int channels, int colors)
{
  unsigned char *d_img_in;
  unsigned char *d_img_out;
  char *h_img_out;
  size_t image_data_size = sizeof(unsigned char)*cols*rows*channels;
  h_img_out = (char *)malloc(image_data_size);
  gpuErrchk(hipMalloc(&d_img_in, image_data_size));
  gpuErrchk(hipMalloc(&d_img_out, image_data_size));
  gpuErrchk(hipMemcpy(d_img_in, image_rgb, image_data_size, hipMemcpyHostToDevice));
  const dim3 blockSize(16,16,1);
  const dim3 gridSize(cols/blockSize.x+1,rows/blockSize.y+1,1);
  mode<<<gridSize, blockSize>>>(d_img_in, d_img_out, cols, rows, channels);
  gpuErrchk(hipFree(d_img_in));
  gpuErrchk(hipMemcpy(h_img_out, d_img_out, image_data_size, hipMemcpyDeviceToHost));
  return h_img_out;
}