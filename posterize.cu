#include "hip/hip_runtime.h"
#include "posterize.h"

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__
void posterize(const unsigned char* input, 
               unsigned char* output, 
               size_t cols, size_t rows, int channels, int n)
{
  int x = blockDim.x*blockIdx.x+threadIdx.x;
  int y = blockDim.y*blockIdx.y+threadIdx.y;
  if (x >= cols || y >= rows) {
      return;
  }
  int idx = y*cols+x;
  int w = 256/n;
  output[idx*channels+0] = (input[idx*channels+0]/w)*w+w/2;
  output[idx*channels+1] = (input[idx*channels+1]/w)*w+w/2;
  output[idx*channels+2] = (input[idx*channels+2]/w)*w+w/2;
}

__global__
void mode(const unsigned char* input,
          unsigned char* output,
          size_t cols, size_t rows, int channels)
{
  int x = blockDim.x*blockIdx.x+threadIdx.x;
  int y = blockDim.y*blockIdx.y+threadIdx.y;
  if (x >= cols || y >= rows) {
      return;
  }

  int idx = y*cols+x;

  int dim = 9;

  int offset, Offset = 0;

  int count = 0, maxCount = 0;

  unsigned char mode = NULL;

  int i, j, k, J, K = 0;

  // for each channel...
  for (i = 0; i < channels; i++) {
    // for every pixel per channel...
    for (j = -dim/2; j <= dim/2; j++){
      count = 0;
      for (k = -dim/2; k <= dim/2; k++) {
        offset = idx+cols*j+k;
        if ((x+k >= cols || y+j >= rows) || (x+k < 0 || y+j < 0)) {
          continue;
        }

        // compare it to every other pixel
        for (J = -dim/2; J <= dim/2; J++){
          for (K = -dim/2; K <= dim/2; K++) {
            Offset = idx+cols*J+K;
            if ((x+K >= cols || y+J >= rows) || (x+K < 0 || y+J < 0)) {
              continue;
            }

            if (input[offset*channels + i] == input[Offset*channels + i]) {
              count++;
            }
          }
        }
        if (count > maxCount) {
          maxCount = count;
          mode = input[offset*channels + i];
        }
      }
    }

    if (maxCount > 1) {
      output[idx*channels + i] = mode;
    }
    else {
      output[idx*channels + i] = input[idx*channels + i];
    }
    maxCount = 0;
  }
}

__global__
void smooth(const unsigned char* input,
            unsigned char* output,
            size_t cols, size_t rows, int channels, int n,
            int *hist)
{
  int x = blockDim.x*blockIdx.x+threadIdx.x;
  int y = blockDim.y*blockIdx.y+threadIdx.y;
  if (x >= cols || y >= rows) {
      return;
  }
  int id = y*cols+x;
  int w = 256/n;
  int i, j, windowSize = 5;
  int idx, idy;
  int size = n*n*n;
  int pixel_id;
  int r, g, b;

  for (i = 0; i < size; i++) {
    hist[i] = 0;   
  }

  int sum_r = 0;
  int sum_g = 0;
  int sum_b = 0;

  for (i = -windowSize/2; i<=windowSize/2; i++) {
    idy = min(max((y + i), 0), (int) rows);
    for (j = -windowSize/2; j<=windowSize/2; j++) {
      idx = min(max((x + j), 0), (int) cols);
      pixel_id = idy*cols + idx;
      r = input[pixel_id*channels+0]/w; sum_r += input[pixel_id*channels+0];
      g = input[pixel_id*channels+1]/w; sum_g += input[pixel_id*channels+1];
      b = input[pixel_id*channels+2]/w; sum_b += input[pixel_id*channels+2];
      hist[r*n*n + g*n + b]++;
    }
  }

  int max = 0;
  int max_index = 0;
  for (i = 0; i<size; i++) {
    if (hist[i] > max) {
      max_index = i;
      max = hist[i];
    }
  }

  // unsigned char mode_r = sum_r / (windowSize*windowSize);
  // unsigned char mode_g = sum_g / (windowSize*windowSize);
  // unsigned char mode_b = sum_b / (windowSize*windowSize);

  unsigned char mode_r = (unsigned char) (max_index/(n*n))*w+w/2;
  unsigned char mode_g = (unsigned char) ((max_index/n)%n)*w+w/2;
  unsigned char mode_b = (unsigned char) (max_index%(n*n))*w+w/2;

  output[id*channels+0] = mode_r;
  output[id*channels+1] = mode_g;
  output[id*channels+2] = mode_b;
}

char* processPosterize(char* image_rgb, size_t cols, size_t rows, int channels, int colors)
{
  unsigned char *d_img_in;
  unsigned char *d_img_out;
  char *h_img_out;
  size_t image_data_size = sizeof(unsigned char)*cols*rows*channels;
  h_img_out = (char *)malloc(image_data_size);
  gpuErrchk(hipMalloc(&d_img_in, image_data_size));
  gpuErrchk(hipMalloc(&d_img_out, image_data_size));
  gpuErrchk(hipMemcpy(d_img_in, image_rgb, image_data_size, hipMemcpyHostToDevice));
  const dim3 blockSize(16,16,1);
  const dim3 gridSize(cols/blockSize.x+1,rows/blockSize.y+1,1);
  // posterize<<<gridSize, blockSize>>>(d_img_in, d_img_out, cols, rows, channels, colors);
  mode<<<gridSize, blockSize>>>(d_img_in, d_img_out, cols, rows, channels);
  gpuErrchk(hipFree(d_img_in));
  gpuErrchk(hipMemcpy(h_img_out, d_img_out, image_data_size, hipMemcpyDeviceToHost));
  return h_img_out;
}
