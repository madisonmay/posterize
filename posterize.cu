#include "hip/hip_runtime.h"
#include "posterize.h"

__global__
void posterize(const unsigned char* input, 
               unsigned char* output, 
               size_t cols, size_t rows, int n)
{
  // int i;
  // int size = cols*rows*3;
  // int w = 256/n;
  // for (i=0; i<size; i++) {
  //   output[i] = (input[i]/w)*w+w/2;
  // }
  int x = blockDim.x*blockIdx.x+threadIdx.x;
  int y = blockDim.y*blockIdx.y+threadIdx.y;
  if (x >= cols || y >= rows) {
      return;
  }
  int idx = x+y*cols;
  output[idx] = input[idx]; //copy image
}